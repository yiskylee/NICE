#include "hip/hip_runtime.h"
// The MIT License (MIT)
//
// Copyright (c) 2016 Northeastern University
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <iostream>
#include "include/kdac_gpu.h"
#include "include/gpu_util.h"

// Hack to cope with Clion
#include "../../include/gpu_util.h"
#include "../../../../../../../../usr/local/cuda/include/hip/driver_types.h"
#include "../../include/kdac_gpu.h"
#include "../../include/kernel_types.h"

namespace Nice {

unsigned int nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

bool isPow2(unsigned int x) {
  return ((x & (x - 1)) == 0);
}

template <typename T>
__device__ void mv(T *mat_s,
                   T* vec_in_s,
                   const int num_rows,
                   const int num_cols,
                   T* vec_out_s) {
  int tx = threadIdx.x;
  int block_size = blockDim.x;
  for (int k = tx; k < num_rows; k += block_size) {
    for (int col = 0; col < num_cols; col++)
      vec_out_s[k] += mat_s[IDXC(k, col, num_rows)] * vec_in_s[col];
  }
  __syncthreads();
}

template <typename T>
__device__ T reduce_sum(T *data_s, int n) {
  T sum = 0;
  int block_size = blockDim.x * blockDim.y;
  int tx = threadIdx.y * blockDim.x + threadIdx.x;

  for (int k = tx; k < n; k += block_size)
    sum += data_s[k];

  data_s[tx] = sum;
  __syncthreads();

  if ((block_size >= 512) && (tx < 256))
    data_s[tx] = sum = sum + data_s[tx + 256];
  __syncthreads();

  if ((block_size >= 256) && (tx < 128))
    data_s[tx] = sum = sum + data_s[tx + 128];
  __syncthreads();

  if ((block_size >= 128) && (tx < 64))
    data_s[tx] = sum = sum + data_s[tx + 64];
  __syncthreads();

//    if ((block_size >= 64) && (tx < 32))
//      exp_term_s[tx] = sum = sum + exp_term_s[tx + 32];
//    __syncthreads();
//
//    if ((block_size >= 32) && (tx < 16))
//      exp_term_s[tx] = sum = sum + exp_term_s[tx + 16];
//    __syncthreads();
//
//    if ((block_size >= 16) && (tx < 8))
//      exp_term_s[tx] = sum = sum + exp_term_s[tx + 8];
//    __syncthreads();
//
//    if ((block_size >= 8) && (tx < 4))
//      exp_term_s[tx] = sum = sum + exp_term_s[tx + 4];
//    __syncthreads();
//
//    if ((block_size >= 4) && (tx < 2))
//      exp_term_s[tx] = sum = sum + exp_term_s[tx + 2];
//    __syncthreads();
//
//    if ((block_size >= 2) && (tx < 1))
//      exp_term_s[tx] = sum = sum + exp_term_s[tx + 1];
//    __syncthreads();

  if (tx < 32) {
    if (block_size >= 64)
      sum += data_s[tx + 32];
    for (int offset = warpSize / 2; offset >0; offset /=2)
      sum += __shfl_down(sum, offset);
  }
  if (tx == 0)
    data_s[tx] = sum;
  __syncthreads();
  return data_s[0];
}


template <typename T>
__device__ void GenAij(const T *x_matrix_d,
                       const int n,
                       const int d,
                       T *a_ij_d,
                       T *delta_ij_d) {
  int tx = threadIdx.x;
  int i = blockIdx.y;
  int j = blockIdx.x;
  int block_size = blockDim.x;

  for (int k = tx; k < d; k += block_size)
    delta_ij_d[k] = x_matrix_d[IDXC(i, k, n)] - x_matrix_d[IDXC(j, k, n)];
    __syncthreads();

  for (int k = tx; k < d; k += block_size)
    for (int col = 0; col < d; col++)
      // thread tx calculates a whole row tx of the output matrix a_ij
      a_ij_d[IDXC(k, col, d)] = delta_ij_d[col] * delta_ij_d[k];
  __syncthreads();
}

template<typename T>
__global__ void UpdateGOfWKernel(const T *x_matrix_d,
                                 const T *w_l_d,
                                 const float constant,
                                 const int n,
                                 const int d,
                                 T *g_of_w_d_) {
  // Shared memory to store a_ij * w_l
  T *delta_w_s = SharedMemory<T>();

  int i = blockIdx.y;
  int j = blockIdx.x;
  int tx = threadIdx.x;
  int block_size = blockDim.x;

  for (int k = tx; k < d; k += block_size)
    delta_w_s[k] =
        (x_matrix_d[IDXC(i,k,n)] - x_matrix_d[IDXC(j,k,n)]) * w_l_d[k];
  __syncthreads();

  T delta_w =reduce_sum(delta_w_s, d);
  T exp_term = expf(-delta_w * delta_w / (2 * constant * constant));
  if (tx == 0)
    g_of_w_d_[IDXC(i,j,n)] *= exp_term;
}

template<typename T>
__global__ void GenPhiCoeffKernel(const T *x_matrix_d,
                                  const T *w_l_d,
                                  const T *gradient_d,
                                  const int n,
                                  const int d,
                                  T *waw_matrix_d,
                                  T *waf_matrix_d,
                                  T *faf_matrix_d) {
  T *delta_w_s = SharedMemory<T>();
  T *delta_f_s = SharedMemory<T>() + d;

  int i = blockIdx.y;
  int j = blockIdx.x;
  int tx = threadIdx.x;
  int block_size = blockDim.x;

  T delta_ij_k;

  for (int k = tx; k < d; k += block_size) {
    delta_ij_k = x_matrix_d[IDXC(i,k,n)] - x_matrix_d[IDXC(j,k,n)];
    delta_w_s[k] = delta_ij_k * w_l_d[k];
    delta_f_s[k] = delta_ij_k * gradient_d[k];
  }
  __syncthreads();

  T delta_w = reduce_sum(delta_w_s, d);
  T delta_f = reduce_sum(delta_f_s, d);

  if (tx == 0) {
    waw_matrix_d[IDXC(j, i, n)] = delta_w * delta_w;
    waf_matrix_d[IDXC(j, i, n)] = delta_w * delta_f;
    faf_matrix_d[IDXC(j, i, n)] = delta_f * delta_f;
  }


//  T *vec_s = SharedMemory<T>();
//  T *waw_s = (T *) vec_s;
//  T *waf_s = (T *) &vec_s[d];
//  T *faf_s = (T *) &vec_s[2 * d];
//  T *w_s = (T *) &vec_s[3 * d];
//  T *grad_s = (T *) &vec_s[4 * d];
//  T *a_ij_s = (T *) &vec_s[5 * d];
//  T *delta_ij_s = (T *) &vec_s[5 * d + d * d];
//
//  GenAij(x_matrix_d, n, d, a_ij_s, delta_ij_s);
//
//  int i = blockIdx.y;
//  int j = blockIdx.x;
//  int tx = threadIdx.x;
//  int block_size = blockDim.x;
//
//  for (int k = tx; k < d; k += block_size) {
//    waw_s[k] = 0.0;
//    waf_s[k] = 0.0;
//    faf_s[k] = 0.0;
//    w_s[k] = w_l_d[k];
//    grad_s[k] = gradient_d[k];
//  }
//  __syncthreads();
//
//  mv(a_ij_s, w_s, d, d, waw_s);
//  mv(a_ij_s, grad_s, d, d, waf_s);
//  mv(a_ij_s, grad_s, d, d, faf_s);
//
//  // Dot Product
//  for (int k = tx; k < d; k += block_size) {
//    waw_s[k] = waw_s[k] * w_s[k];
//    waf_s[k] = waf_s[k] * w_s[k];
//    faf_s[k] = faf_s[k] * grad_s[k];
//  }
//  __syncthreads();
//  reduce_sum(waw_s, d);
//  reduce_sum(waf_s, d);
//  reduce_sum(faf_s, d);
//
//  // Transposed access for better access pattern as waw_s matrix is column-major
//  if (tx == 0) {
//    waw_matrix_d[IDXC(j, i, n)] = waw_s[tx];
//    waf_matrix_d[IDXC(j, i, n)] = waf_s[tx];
//    faf_matrix_d[IDXC(j, i, n)] = faf_s[tx];
//  }


}

template<typename T>
__global__ void GenPhiKernel(const float alpha,
                             const float sqrt_one_minus_alpha,
                             const float denom,
                             const T *waw_matrix_d,
                             const T *waf_matrix_d,
                             const T *faf_matrix_d,
                             const T *gamma_matrix_d,
                             const int n,
                             const int d,
                             bool w_l_changed,
                             T *phi_of_alphas_d,
                             T *phi_of_zeros_d,
                             T *phi_of_zero_primes_d) {

  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int block_size = blockDim.x * blockDim.y;
  int tid = IDXR(threadIdx.y, threadIdx.x, blockDim.x);
  int bid = IDXR(blockIdx.y, blockIdx.x, gridDim.x);


  T *phi_of_alphas_s = SharedMemory<T>();
  T *phi_of_zeros_s = 0;
  T *phi_of_zero_primes_s = 0;

  phi_of_alphas_s[tid] = 0.0;
  if (w_l_changed) {
    phi_of_zeros_s = SharedMemory<T>() + block_size;
    phi_of_zero_primes_s = SharedMemory<T>() + 2*block_size;
    phi_of_zeros_s[tid] = 0.0;
    phi_of_zero_primes_s[tid] = 0.0;
  }
  __syncthreads();

  if ((i < n) && (j < n)) {
    T waw = waw_matrix_d[IDXC(j, i, n)];
    T waf = waf_matrix_d[IDXC(j, i, n)];
    T faf = faf_matrix_d[IDXC(j, i, n)];
    T gammaij = gamma_matrix_d[IDXC(j, i, n)];
    T kij = expf(denom * ((faf - waw) * (alpha * alpha) +
        2 * waf * sqrt_one_minus_alpha * alpha + waw));
    phi_of_alphas_s[tid] = gammaij * kij;
    if (w_l_changed) {
      T kij = expf(denom * waw);
      phi_of_zeros_s[tid] = gammaij * kij;
      phi_of_zero_primes_s[tid] = gammaij * denom * 2 * waf * kij;
//    phi_of_alphas_d[IDXC(j, i, n)] = gammaij * kij;
    }
    __syncthreads();

    T phi_of_alpha = reduce_sum(phi_of_alphas_s, block_size);
    T phi_of_zero = 0;
    T phi_of_zero_prime = 0;

    if (w_l_changed) {
      phi_of_zero = reduce_sum(phi_of_zeros_s, block_size);
      phi_of_zero_prime = reduce_sum(phi_of_zero_primes_s, block_size);
    }

    if (tid == 0) {
      phi_of_alphas_d[bid] = phi_of_alpha;
      if (w_l_changed) {
        phi_of_zeros_d[bid] = phi_of_zero;
        phi_of_zero_primes_d[bid] = phi_of_zero_prime;
      }
    }


//    for (unsigned int s = (blockDim.x * blockDim.y / 2); s > 0; s >>= 1) {
//      if (tid < s) {
//        phi_of_alphas_s[tid] += phi_of_alphas_s[tid + s];
//        if (w_l_changed) {
//          phi_of_zeros_s[tid] += phi_of_zeros_s[tid + s];
//          phi_of_zero_primes_s[tid] +=
//              phi_of_zero_primes_s[tid + s];
//        }
//      }
//      __syncthreads();
//    }
//    if (tid == 0) {
//      phi_of_alphas_d[bid] = phi_of_alphas_s[tid];
//      if (w_l_changed) {
//        phi_of_zeros_d[bid] = phi_of_zeros_s[tid];
//        phi_of_zero_primes_d[bid] = phi_of_zero_primes_s[tid];
//      }
//    }
  }
}

template<typename T>
__global__ void GenWGradientKernel(const T *x_matrix_d,
                                   const T *g_of_w_d,
                                   const T *w_l_d,
                                   const T *gamma_matrix_d,
                                   const float constant,
                                   const int n,
                                   const int d,
                                   T *gradient_fs_d) {

  T *delta_ij_s = SharedMemory<T>();
  T *delta_w_s = SharedMemory<T>() + d;
  int i = blockIdx.y;
  int j = blockIdx.x;
  int tx = threadIdx.x;
  int block_size = blockDim.x;


  for (int k = tx; k < d; k += block_size) {
    delta_ij_s[k] = x_matrix_d[IDXC(i, k, n)] - x_matrix_d[IDXC(j, k, n)];
    // Dot product for delta' * w
    delta_w_s[k] = delta_ij_s[k] * w_l_d[k];
  }
  __syncthreads();

  T delta_w = reduce_sum(delta_w_s, d);
  T waw = delta_w * delta_w;

  T sigma_sq = constant * constant;

  int index_ij = IDXC(i, j, n);
  T gamma_ij = gamma_matrix_d[index_ij];
  T g_of_w_ij = g_of_w_d[index_ij];
  T exp_term = expf(-waw / (2 * sigma_sq));
  T coeff = -gamma_ij * g_of_w_ij * exp_term / sigma_sq;
  T *gradient_f_ij = gradient_fs_d + IDXR(i, j, n) * d;
  // delta * delta_w == Aij * w
  for (int k = tx; k < d; k += block_size)
    gradient_f_ij[k] = coeff * delta_ij_s[k] * delta_w;
}

template<typename T>
void KDACGPU<T>::GenPhiCoeff(const Vector <T> &w_l,
                             const Vector <T> &gradient) {
  // Three terms used to calculate phi of alpha
  // They only change if w_l or gradient change
  CUDA_CALL(hipMemcpy(w_l_d_, &w_l(0), d_ * sizeof(T),
                       hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(gradient_d_, &gradient(0), d_ * sizeof(T),
                       hipMemcpyHostToDevice));

  unsigned int block_size = (d_ < block_limit_ * 2) ?
                            nextPow2((d_+1)/2) : block_limit_;

  int shared_mem_size = 2 * d_ * sizeof(T);
  dim3 dim_block(block_size, 1);
  dim3 dim_grid(n_, n_);
  GenPhiCoeffKernel <<<dim_grid, dim_block, shared_mem_size>>> (
      x_matrix_d_,
      w_l_d_,
      gradient_d_,
      n_,
      d_,
      waw_matrix_d_,
      waf_matrix_d_,
      faf_matrix_d_);
  CUDA_CALL(hipGetLastError());
}

template
void KDACGPU<float>::GenPhiCoeff(const Vector<float> &w_l,
                                 const Vector<float> &gradient);
template
void KDACGPU<double>::GenPhiCoeff(const Vector<double> &w_l,
                                 const Vector<double> &gradient);

// Generate phi(alpha), phi(0) and phi'(0) for LineSearch
// If this is the first time to generate phi(), then w_l_changed is true
// Or if the w_l is negated because phi'(0) is negative,
// then w_l_changed is true
// If w_l_changed is true, generate phi(0) and phi'(0), otherwise
// when we are only computing phi(alpha) with a different alpha in the loop
// of the LineSearch, the w_l_changed is false and we do not generate
// new waw, waf and faf
template<typename T>
void KDACGPU<T>::GenPhi(const Vector <T> &w_l,
                        const Vector <T> &gradient,
                        bool w_l_changed) {

  if (kernel_type_ == kGaussianKernel) {
    profiler_["gen_phi"].Start();
    float alpha_square = pow(alpha_, 2);
    float sqrt_one_minus_alpha = pow((1 - alpha_square), 0.5);
    float denom = -1 / (2 * pow(constant_, 2));

    phi_of_alpha_ = 0;
    if (w_l_changed) {
      GenPhiCoeff(w_l, gradient);
      phi_of_zero_ = 0;
      phi_of_zero_prime_ = 0;
    }

    int block_dim_x = 16;
    int block_dim_y = 16;
    dim3 dim_block(block_dim_x, block_dim_y);
    // If matrix is n x m, then I need an m x n grid for contiguous
    // memory access
    dim3 dim_grid((n_ - 1) / block_dim_x + 1,
                  (n_ - 1) / block_dim_y + 1);
    int block_size = block_dim_x * block_dim_y;
    int num_blocks =
        ((n_ - 1) / block_dim_x + 1) * ((n_ - 1) / block_dim_y + 1);
    int shared_mem_size;
    if (w_l_changed)
      shared_mem_size = 3 * block_size * sizeof(T);
    else
      shared_mem_size = block_size * sizeof(T);

    GenPhiKernel << < dim_grid, dim_block, shared_mem_size >> >
        (alpha_,
        sqrt_one_minus_alpha,
        denom,
        waw_matrix_d_,
        waf_matrix_d_,
        faf_matrix_d_,
        gamma_matrix_d_,
        n_,
        d_,
        w_l_changed,
        phi_of_alphas_d_,
        phi_of_zeros_d_,
        phi_of_zero_primes_d_);

    // Check if error happens in kernel launch
    CUDA_CALL(hipGetLastError());

    CUDA_CALL(hipMemcpy(phi_of_alphas_h_, phi_of_alphas_d_,
                         num_blocks * sizeof(T), hipMemcpyDeviceToHost));

    for (int i = 0; i < num_blocks; i++) {
      phi_of_alpha_ += phi_of_alphas_h_[i];
    }
    if (w_l_changed) {
      CUDA_CALL(hipMemcpy(phi_of_zeros_h_, phi_of_zeros_d_,
                           num_blocks * sizeof(T), hipMemcpyDeviceToHost));
      CUDA_CALL(hipMemcpy(phi_of_zero_primes_h_, phi_of_zero_primes_d_,
                           num_blocks * sizeof(T), hipMemcpyDeviceToHost));
      for (int i = 0; i < num_blocks; i++) {
        phi_of_zero_ += phi_of_zeros_h_[i];
        phi_of_zero_prime_ += phi_of_zero_primes_h_[i];
      }
    }
    profiler_["gen_phi"].Record();
  }
}

template
void KDACGPU<float>::GenPhi(const Vector<float> &w_l,
                            const Vector<float> &gradient,
                            bool w_l_changed);

template
void KDACGPU<double>::GenPhi(const Vector<double> &w_l,
                            const Vector<double> &gradient,
                            bool w_l_changed);


template<typename T>
Vector<T> KDACGPU<T>::GenWGradient(const Vector <T> &w_l) {
  Vector<T> w_gradient = Vector<T>::Zero(d_);
  if (kernel_type_ == kGaussianKernel) {
    gpu_util_->EigenToDevBuffer(w_l_d_, w_l);

//    CUDA_CALL(hipMemcpy(w_l_d_, &w_l(0), d_ * sizeof(T),
//                         hipMemcpyHostToDevice));
    // When block_limit is 512
    // If d is 128, block_size is 64
    // If d is 6, block_size is 4
    // If d is 1025, block_size is 512
    unsigned int block_size = (d_ < block_limit_ * 2) ?
                              nextPow2((d_+1)/2) : block_limit_;

    int shared_mem_size = 2 * d_ * sizeof(T);

    dim3 dim_block(block_size, 1);
    dim3 dim_grid(n_, n_);
    GenWGradientKernel
        << < dim_grid, dim_block, shared_mem_size >> >
        (x_matrix_d_,
            g_of_w_d_,
            w_l_d_,
            gamma_matrix_d_,
            constant_,
            n_,
            d_,
            grad_f_arr_d_);
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipMemcpy(grad_f_arr_h_, grad_f_arr_d_,
                         n_ * n_ * d_ * sizeof(T),
                         hipMemcpyDeviceToHost));


    for (int i = 0; i < n_; i++) {
      for (int j = 0; j < n_; j++) {
        T *grad_f_ij = grad_f_arr_h_ + IDXR(i, j, n_) * d_;
        Vector<T> grad_temp = Eigen::Map < Vector < T >> (grad_f_ij, d_);
        util::CheckFinite(grad_temp, "grad_temp_"+std::to_string(i));
        w_gradient = w_gradient + grad_temp;
      }
    }
  }
  util::CheckFinite(w_gradient, "w_gradient");
  return w_gradient;
}

template
Vector<float> KDACGPU<float>::GenWGradient(const Vector<float> &w_l);
template
Vector<double> KDACGPU<double>::GenWGradient(const Vector<double> &w_l);


template<typename T>
void KDACGPU<T>::UpdateGOfW(const Vector<T> &w_l) {
  profiler_["update_g_of_w"].Start();
  CUDA_CALL(hipMemcpy(w_l_d_, &w_l(0), d_ * sizeof(T),
                       hipMemcpyHostToDevice));
  if (kernel_type_ == kGaussianKernel) {
    unsigned int block_size = (d_ < block_limit_ * 2) ?
        nextPow2((d_+1)/2) : block_limit_;
    int shared_mem_size = d_ * sizeof(T);
    dim3 dim_block(block_size, 1);
    dim3 dim_grid(n_, n_);
    UpdateGOfWKernel <<<dim_grid, dim_block, shared_mem_size>>>
        (x_matrix_d_,
         w_l_d_,
         constant_,
         n_,
         d_,
         g_of_w_d_);
    CUDA_CALL(hipGetLastError());
  }
  profiler_["update_g_of_w"].Record();
}

template
void KDACGPU<float>::UpdateGOfW(const Vector<float> &w_l);
template
void KDACGPU<double>::UpdateGOfW(const Vector<double> &w_l);

template<typename T>
T KDACGPU<T>::GenPhiOfAlpha(const Vector<T> &w_l) {
  profiler_["gen_phi(alpha)"].Start();
  profiler_["gen_phi(alpha)"].Record();
  return 1.0;
}
template float KDACGPU<float>::GenPhiOfAlpha(const Vector<float> &w_l);
template double KDACGPU<double>::GenPhiOfAlpha(const Vector<double> &w_l);

}  // Namespace NICE